#include "hip/hip_runtime.h"
#include <iostream>

#include <tinytest.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <random>

#include "glm/vec2.hpp"
#include "glm/glm.hpp"
#include "glm/gtc/type_ptr.hpp"

#include "../src/gpu/gpuErrCheck.h"
#include "../src/logging/Logging.h"
#include "../src/timing/timing_helpers.h"
#include "../src/data_structures/mcleap_wrapper.h"

struct SaveNeighborsFunctor {
	SaveNeighborsFunctor(float rad, int numP, int max_neighbors) : m_numP(numP), h_m_max_neighbors(max_neighbors) {
		hipMalloc((void**)&m_rad, sizeof(float));
		hipMalloc((void**)&m_max_neighbors, sizeof(int));
		hipMalloc((void**)&m_num_neighbors, numP * sizeof(int));
		hipMalloc((void**)&m_neighbors, h_m_max_neighbors * numP * sizeof(int));

		hipMemcpy(m_rad, &rad, sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(m_max_neighbors, &max_neighbors, sizeof(int), hipMemcpyHostToDevice);
		hipDeviceSynchronize();
	}
	~SaveNeighborsFunctor() {
		//hipFree(m_rad);
		//hipFree(m_max_neighbors);
		//hipFree(m_num_neighbors);
		//hipFree(m_neighbors);
	}
	void resetFunctor() {
		hipMemset(m_num_neighbors, 0, m_numP * sizeof(int));
		hipMemset(m_neighbors, 0, m_numP * h_m_max_neighbors * sizeof(int));
		hipDeviceSynchronize();
	}
	__device__ void operator()(const int& i, const int& j, const glm::dvec2 dist_vec, const double dist) {
		if (i != j)if (dist <= *m_rad && dist > 0) {
			if(i==4)printf("i: %d j: %d dist: %f\n", i, j, dist);
			int ind = i * (*m_max_neighbors) + m_num_neighbors[i];
			if (ind < m_numP * (*m_max_neighbors))m_neighbors[ind] = j;
			m_num_neighbors[i]++;
		}
	}
	int m_numP, h_m_max_neighbors;
	float* m_rad;
	int* m_max_neighbors;
	int* m_num_neighbors;
	int* m_neighbors;
};

void lattice_test() {
	int numP = 10000;
	int max_neighs = 30;
	float rad = 1.0;

	triangulation2d<100, 100> gc(numP);

	SaveNeighborsFunctor* snfunctor = new SaveNeighborsFunctor(rad, numP, max_neighs);
	snfunctor->resetFunctor();

	glm::dvec2* pos = new glm::dvec2[numP];
	for (int x = 0; x < 100; x++) {
		for (int y = 0; y < 100; y++) {
			int i = x + 100 * +y;
			if (i > numP)continue;
			pos[i] = glm::dvec2(x, y);
		}
	}
	pos[0].x = -1000.0;
	pos[0].y = -1000.0;
	pos[1].x = 1000.0;
	pos[1].y = -1000.0;
	pos[2].x = 1000.0;
	pos[2].y = 1000.0;
	pos[3].x = -1000.0;
	pos[3].y = 1000.0;

	gc.build(pos);

	glm::dvec2* d_pos;
	hipMalloc((void**)&d_pos, numP * sizeof(glm::dvec2));
	hipMemcpy(d_pos, pos, numP * sizeof(glm::dvec2), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	//gc.update((MCleap::MCLEAP_VEC*)d_pos);
	hipDeviceSynchronize();
	gc.apply_f_frnn<SaveNeighborsFunctor>(*snfunctor, d_pos, rad);
	hipDeviceSynchronize();

	int* h_num_neighbors = new int[numP];
	int* h_neighbors = new int[numP * max_neighs];

	hipMemcpy(h_num_neighbors, snfunctor->m_num_neighbors, numP * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_neighbors, snfunctor->m_neighbors, max_neighs * numP * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(pos, gc.m->d_vbo_v, numP * sizeof(glm::dvec2), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();



	int* res_num_neighs = new int[numP];

	for (int i = 0; i < numP; i++) {
		res_num_neighs[i] = 0;
		for (int j = 0; j < numP; j++) {
			if (i == j)continue;
			glm::dvec2 dist_vec = pos[i] - pos[j];
			float dist = glm::dot(dist_vec, dist_vec);
			//if (i == 3)printf("i: %d j: %d dist: %f\n", i, j, sqrt(dist));
			if (dist <= rad * rad) {
				res_num_neighs[i]++;
			}
		}
	}

	for (int i = 0; i < numP; i++) {
		//printf("i: %d num_true: %d num_calc: %d\n", i, res_num_neighs[i], h_num_neighbors[i]);
		ASSERT_EQUALS(res_num_neighs[i], h_num_neighbors[i]);
	}

	delete[] res_num_neighs;

	delete[] pos;

	delete[] h_num_neighbors;
	delete[] h_neighbors;
	delete snfunctor;
	hipFree(d_pos);
}

void packed_lattice_test() {
	int numP = 400;
	int max_neighs = 40;
	float rad = 1.0;
	triangulation2d<100, 100> gc(numP);

	SaveNeighborsFunctor* snfunctor = new SaveNeighborsFunctor(rad, numP, max_neighs);
	snfunctor->resetFunctor();

	glm::dvec2* pos = new glm::dvec2[numP];
	for (int x = 0; x < 20; x++) {
		for (int y = 0; y < 20; y++) {
			int i = x + 20 * y;
			if (i > numP)continue;
			pos[i] = glm::dvec2(x * 0.5, y * 0.5);
		}
	}
	pos[0].x = -1000.0;
	pos[0].y = -1000.0;
	pos[1].x = 1000.0;
	pos[1].y = -1000.0;
	pos[2].x = 1000.0;
	pos[2].y = 1000.0;
	pos[3].x = -1000.0;
	pos[3].y = 1000.0;

	gc.build(pos);

	glm::dvec2* d_pos;
	hipMalloc((void**)&d_pos, numP * sizeof(glm::dvec2));
	hipMemcpy(d_pos, pos, numP * sizeof(glm::dvec2), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	//gc.update(d_pos);
	hipDeviceSynchronize();
	gc.apply_f_frnn<SaveNeighborsFunctor>(*snfunctor, d_pos, rad);
	hipDeviceSynchronize();

	int* h_num_neighbors = new int[numP];
	int* h_neighbors = new int[numP * max_neighs];

	hipMemcpy(h_num_neighbors, snfunctor->m_num_neighbors, numP * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_neighbors, snfunctor->m_neighbors, max_neighs * numP * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(pos, d_pos, numP * sizeof(glm::dvec2), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	int* res_num_neighs = new int[numP];

	for (int i = 0; i < numP; i++) {
		res_num_neighs[i] = 0;
		for (int j = 0; j < numP; j++) {
			if (i == j)continue;
			glm::dvec2 dist_vec = pos[i] - pos[j];
			float dist = glm::dot(dist_vec, dist_vec);
			if (dist <= rad * rad) {
				if(i==4)printf("i: %d j: %d pos[i]: %f %f pos[j]: %f %f dist: %f\n", i, j, pos[i].x, pos[i].y, pos[j].x, pos[j].y, sqrt(dist));
				res_num_neighs[i]++;
			}
		}

		printf("i: %d num_true: %d num_calc: %d\n", i, res_num_neighs[i], h_num_neighbors[i]);
		ASSERT_EQUALS(res_num_neighs[i], h_num_neighbors[i]);
	}

	delete[] res_num_neighs;
	delete[] pos;

	delete[] h_num_neighbors;
	delete[] h_neighbors;
	delete snfunctor;
	hipFree(d_pos);
}
//
//void random_points_test() {
//	int numP = 10000;
//	int max_neighs = 40;
//	float rad = 1.0;
//	glm::dvec2 min(-50.0);
//	glm::dvec2 cell_size(1.0);
//	glm::ivec2 num_cells(100);
//	GridCount2d gc(numP, min, cell_size, num_cells);
//
//	SaveNeighborsFunctor* snfunctor = new SaveNeighborsFunctor(rad, numP, max_neighs);
//	snfunctor->resetFunctor();
//
//	std::random_device dev;
//	std::mt19937 rng{ dev() };
//	rng.seed(10);
//	std::uniform_real_distribution<> dista(-50.0, 50.0);
//
//	glm::dvec2* pos = new glm::dvec2[numP];
//	for (int i = 0; i < numP; i++) {
//		pos[i] = glm::dvec2(dista(rng), dista(rng));
//	}
//
//	glm::dvec2* d_pos;
//	hipMalloc((void**)&d_pos, numP * sizeof(glm::dvec2));
//	hipMemcpy(d_pos, pos, numP * sizeof(glm::dvec2), hipMemcpyHostToDevice);
//	hipDeviceSynchronize();
//
//	gc.update(d_pos);
//	hipDeviceSynchronize();
//	gc.apply_f_frnn<SaveNeighborsFunctor>(*snfunctor, d_pos, rad);
//	hipDeviceSynchronize();
//
//	int* h_num_neighbors = new int[numP];
//	int* h_neighbors = new int[numP * max_neighs];
//
//	hipMemcpy(h_num_neighbors, snfunctor->m_num_neighbors, numP * sizeof(int), hipMemcpyDeviceToHost);
//	hipMemcpy(h_neighbors, snfunctor->m_neighbors, max_neighs * numP * sizeof(int), hipMemcpyDeviceToHost);
//	hipMemcpy(pos, d_pos, numP * sizeof(glm::dvec2), hipMemcpyDeviceToHost);
//	hipDeviceSynchronize();
//
//	int* res_num_neighs = new int[numP];
//
//	for (int i = 0; i < numP; i++) {
//		if (i % 1000)printProgress((double)i * (1.0 / (double)numP));
//
//
//		res_num_neighs[i] = 0;
//		for (int j = 0; j < numP; j++) {
//			if (i == j)continue;
//			glm::dvec2 dist_vec = pos[i] - pos[j];
//			float dist = glm::dot(dist_vec, dist_vec);
//			const double EPS = 0.0001;
//			if ((dist - EPS) <= (rad * rad) || (sqrt(dist - EPS) <= rad)) {
//				//printf("i: %d j: %d hi: %d hj: %d pos[i]: %f %f %f pos[j]: %f %f %f dist: %f\n", i, j, calcHash(pos[i], gc.h_gcdata), calcHash(pos[j], gc.h_gcdata), pos[i].x, pos[i].y, pos[i].z, pos[j].x, pos[j].y, pos[j].z, sqrt(dist));
//				res_num_neighs[i]++;
//			}
//		}
//
//		//printf("i: %d num_neighs: %d h_num_neighs: %d pos[i]: % f %f %f\n", i, res_num_neighs[i], h_num_neighbors[i], pos[i].x, pos[i].y, pos[i].z);
//		for (int j = 0; j < h_num_neighbors[i]; j++) {
//			double leng = glm::length(pos[h_neighbors[i * max_neighs + j]] - pos[i]);
//			//printf("j: %d pos: %f %f %f dist: %f\n", h_neighbors[i * max_neighs + j], pos[h_neighbors[i * max_neighs + j]].x, pos[h_neighbors[i * max_neighs + j]].y, pos[h_neighbors[i * max_neighs + j]].z, leng);
//		}
//
//		if (abs(res_num_neighs[i] - h_num_neighbors[i]) > 1)printf("i: %d num_neighs: %d h_num_neighs: %d pos[i]: % f %f\n", i, res_num_neighs[i], h_num_neighbors[i], pos[i].x, pos[i].y);
//		ASSERT_TRUE(abs(res_num_neighs[i] - h_num_neighbors[i]) <= 1);
//	}
//
//	delete[] res_num_neighs;
//	delete[] pos;
//
//	delete[] h_num_neighbors;
//	delete[] h_neighbors;
//	delete snfunctor;
//	hipFree(d_pos);
//}
//
//void more_packed_lattice_test() {
//	int numP = 64000;
//	int max_neighs = 10;
//	float rad = 0.25;
//	glm::dvec2 min(0.0);
//	glm::dvec2 cell_size(1.0);
//	glm::ivec2 num_cells(10);
//	GridCount2d gc(numP, min, cell_size, num_cells);
//
//	SaveNeighborsFunctor* snfunctor = new SaveNeighborsFunctor(rad, numP, max_neighs);
//	snfunctor->resetFunctor();
//	hipDeviceSynchronize();
//	gpuErrchk(hipGetLastError());
//
//	glm::dvec2* pos = new glm::dvec2[numP];
//	for (int x = 0; x < 40; x++) {
//		for (int y = 0; y < 40; y++) {
//			for (int z = 0; z < 40; z++) {
//				int i = x + 40 * y;
//				if (i > numP)continue;
//				pos[i] = glm::dvec2(x * 0.25, y * 0.25);
//			}
//		}
//	}
//
//	glm::dvec2* d_pos;
//	hipMalloc((void**)&d_pos, numP * sizeof(glm::dvec2));
//	hipMemcpy(d_pos, pos, numP * sizeof(glm::dvec2), hipMemcpyHostToDevice);
//	hipDeviceSynchronize();
//	gpuErrchk(hipGetLastError());
//
//	gc.update(d_pos);
//	hipDeviceSynchronize();
//	gc.apply_f_frnn<SaveNeighborsFunctor>(*snfunctor, d_pos, rad);
//	hipDeviceSynchronize();
//	gpuErrchk(hipGetLastError());
//
//	int* h_num_neighbors = new int[numP];
//	int* h_neighbors = new int[numP * max_neighs];
//
//	gpuErrchk(hipMemcpy(h_num_neighbors, snfunctor->m_num_neighbors, numP * sizeof(int), hipMemcpyDeviceToHost));
//	gpuErrchk(hipMemcpy(h_neighbors, snfunctor->m_neighbors, max_neighs * numP * sizeof(int), hipMemcpyDeviceToHost));
//	gpuErrchk(hipMemcpy(pos, d_pos, numP * sizeof(glm::dvec2), hipMemcpyDeviceToHost));
//	hipDeviceSynchronize();
//	gpuErrchk(hipGetLastError());
//
//	int* res_num_neighs = new int[numP];
//
//	for (int i = 0; i < numP; i++) {
//		printProgress((double)i / (double)numP);
//		res_num_neighs[i] = 0;
//		for (int j = 0; j < numP; j++) {
//			if (i == j)continue;
//			glm::dvec2 dist_vec = pos[i] - pos[j];
//			float dist = glm::dot(dist_vec, dist_vec);
//			if (dist <= rad * rad || sqrt(dist) <= rad) {
//				//printf("i: %d j: %d hi: %d hj: %d pos[i]: %f %f %f pos[j]: %f %f %f dist: %f\n", i, j, calcHash(pos[i], gc.h_gcdata), calcHash(pos[j], gc.h_gcdata), pos[i].x, pos[i].y, pos[i].z, pos[j].x, pos[j].y, pos[j].z, sqrt(dist));
//				res_num_neighs[i]++;
//			}
//		}
//
//		//printf("i: %d num_neighs: %d h_num_neighs: %d pos[i]: % f %f %f\n", i, res_num_neighs[i], h_num_neighbors[i], pos[i].x, pos[i].y, pos[i].z);
//		//for (int j = 0; j < h_num_neighbors[i]; j++) {
//		//	double leng = glm::length(pos[h_neighbors[i * max_neighs + j]] - pos[i]);
//		//	printf("j: %d pos: %f %f %f dist: %f\n", h_neighbors[i * max_neighs + j], pos[h_neighbors[i * max_neighs + j]].x, pos[h_neighbors[i * max_neighs + j]].y, pos[h_neighbors[i * max_neighs + j]].z, leng);
//		//}
//
//		ASSERT_EQUALS(res_num_neighs[i], h_num_neighbors[i]);
//	}
//
//	delete[] res_num_neighs;
//	delete[] pos;
//
//	delete[] h_num_neighbors;
//	delete[] h_neighbors;
//	delete snfunctor;
//	hipFree(d_pos);
//}

int main() {
	init_logging();
	RUN(lattice_test);
	printf("\n");
	RUN(packed_lattice_test);
//	RUN(more_packed_lattice_test);
//	RUN(random_points_test);
	return TEST_REPORT();
}
