#include "hip/hip_runtime.h"
#include <iostream>

#include <tinytest.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "glm/vec3.hpp"
#include "glm/glm.hpp"

#include "../include/gpuErrCheck.h"
#include "../include/Logging.h"
#include "../include/timing_helpers.h"
#include "../grid2.h"

struct SaveNeighborsFunctor {
	SaveNeighborsFunctor(float rad, int numP, int max_neighbors) : m_numP(numP) {
		hipMalloc((void**)&m_rad,		   sizeof(float));
		hipMalloc((void**)&m_max_neighbors, sizeof(int));
		hipMalloc((void**)&m_num_neighbors, numP*sizeof(int));
		hipMalloc((void**)&m_neighbors,     numP*sizeof(int));
		
		hipMemcpy(m_rad, &rad, sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(m_max_neighbors, &max_neighbors, sizeof(int), hipMemcpyHostToDevice);
		hipDeviceSynchronize();

	}
	~SaveNeighborsFunctor() {
		hipFree(m_rad);
		hipFree(m_max_neighbors);
		hipFree(m_num_neighbors);
		hipFree(m_neighbors);
	}
	void resetFunctor() {
		hipMemset(m_num_neighbors, 0, m_numP * sizeof(int));
		hipMemset(m_neighbors,     0, m_numP * sizeof(int));
	}
	__device__ void operator()(const int& i, const int& j, const glm::vec3 dist_vec, const double dist) {
		printf("i: %d j: %d\n", i, j);
		if(i!=j)if (dist <= *m_rad)m_neighbors[i * (*m_max_neighbors) + atomicAdd(&m_num_neighbors[i],1)] = j;
	}
	int m_numP;
	float* m_rad;
	int* m_max_neighbors;
	int* m_num_neighbors;
	int* m_neighbors;
};

void test_neighbors() {
	int numP = 1000;
	glm::vec3 min(0.0);
	glm::vec3 cell_size(1.0);
	glm::ivec3 num_cells(10);
	GridCount gc(numP, min, cell_size, num_cells);

	SaveNeighborsFunctor snfunctor(1.0, numP, 20);
	snfunctor.resetFunctor();

	glm::vec3* pos = new glm::vec3[numP];
	for (int x = 0; x < 10; x++) {
		for (int y = 0; y < 10; y++) {
			for (int z = 0; z < 10; z++) {
				int i = x + 10 * (z * 10 + y);
				pos[i] = glm::vec3((x%10)*0.5, (y%10)*0.5, (z%10)*0.5);
				printf("x: %d y: %d z: %d pos[i]: %f %f %f\n", x, y, z, pos[i].x, pos[i].y, pos[i].z);
			}
		}
	}
	glm::vec3* d_pos;
	hipMalloc((void**)&d_pos, numP*sizeof(glm::vec3));
	hipMemcpy(d_pos, &pos, numP*sizeof(glm::vec3), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	gc.apply_f_frnn<SaveNeighborsFunctor>(snfunctor, d_pos, 2.0);

}


int main() {
	init_logging();
	RUN(test_neighbors);
	return TEST_REPORT();
}
