#include "hip/hip_runtime.h"
#include <iostream>

#include <tinytest.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "glm/vec3.hpp"
#include "glm/glm.hpp"

#include "../include/gpuErrCheck.h"
#include "../include/Logging.h"
#include "../include/timing_helpers.h"
#include "../grid2.h"

struct SaveNeighborsFunctor {
	SaveNeighborsFunctor(float rad, int numP, int max_neighbors) : m_numP(numP), h_m_max_neighbors(max_neighbors) {
		hipMalloc((void**)&m_rad,		   sizeof(float));
		hipMalloc((void**)&m_max_neighbors, sizeof(int));
		hipMalloc((void**)&m_num_neighbors, numP*sizeof(int));
		hipMalloc((void**)&m_neighbors, h_m_max_neighbors*numP*sizeof(int));
		
		hipMemcpy(m_rad, &rad, sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(m_max_neighbors, &max_neighbors, sizeof(int), hipMemcpyHostToDevice);
		hipDeviceSynchronize();
	}
	~SaveNeighborsFunctor() = default; // left a memory leak on porpouse, because for some reason, this program calls the destructor way before it should be
	void resetFunctor() {
		hipMemset(m_num_neighbors, 0, m_numP * sizeof(int));
		hipMemset(m_neighbors,     0, m_numP * h_m_max_neighbors * sizeof(int));
		hipDeviceSynchronize();
	}
	__device__ void operator()(const int& i, const int& j, const glm::vec3 dist_vec, const double dist) {
		if (i != j)if (dist <= *m_rad) {
			int ind = i * (*m_max_neighbors) + m_num_neighbors[i];
			m_neighbors[ind] = j;
			m_num_neighbors[i]++;
		}
	}
	int m_numP, h_m_max_neighbors;
	float* m_rad;
	int* m_max_neighbors;
	int* m_num_neighbors;
	int* m_neighbors;
};

void lattice_test() {
	int numP = 1000;
	int max_neighs = 30;
	float rad = 1.0;
	glm::vec3 min(0.0);
	glm::vec3 cell_size(1.0);
	glm::ivec3 num_cells(10);
	GridCount gc(numP, min, cell_size, num_cells);

	SaveNeighborsFunctor* snfunctor = new SaveNeighborsFunctor(rad, numP, max_neighs);
	snfunctor->resetFunctor();

	glm::vec3* pos = new glm::vec3[numP];
	for (int x = 0; x < 10; x++) {
		for (int y = 0; y < 10; y++) {
			for (int z = 0; z < 10; z++) {
				int i = x + 10 * (z * 10 + y);
				if (i > numP)continue;
				pos[i] = glm::vec3(x, y, z);
			}
		}
	}

	glm::vec3* d_pos;
	hipMalloc((void**)&d_pos, numP*sizeof(glm::vec3));
	hipMemcpy(d_pos, pos, numP*sizeof(glm::vec3), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	gc.update(d_pos);
	hipDeviceSynchronize();
	gc.apply_f_frnn<SaveNeighborsFunctor>(*snfunctor, d_pos, rad);
	hipDeviceSynchronize();

	int* h_num_neighbors = new int[numP];
	int* h_neighbors = new int[numP * max_neighs];
	
	hipMemcpy(h_num_neighbors, snfunctor->m_num_neighbors, numP * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_neighbors, snfunctor->m_neighbors, max_neighs* numP * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(pos, d_pos, numP * sizeof(glm::vec3), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	int* res_num_neighs = new int[numP];

	for (int i = 0; i < numP; i++) {
		res_num_neighs[i] = 0;
		for (int j = 0; j < numP; j++) {
			if (i == j)continue;
			glm::vec3 dist_vec = pos[i] - pos[j];
			float dist = glm::dot(dist_vec, dist_vec);
			if (dist <= rad * rad) {
				res_num_neighs[i]++;
			}
		}
	}

	for (int i = 0; i < numP; i++) {
		ASSERT_EQUALS(res_num_neighs[i], h_num_neighbors[i]);
	}

	delete[] res_num_neighs;
	
	delete[] pos;

	delete[] h_num_neighbors;
	delete[] h_neighbors;
	delete snfunctor;
	hipFree(d_pos);
}

void packed_lattice_test() {
	int numP = 8000;
	int max_neighs = 40;
	float rad = 1.0;
	glm::vec3 min(0.0);
	glm::vec3 cell_size(4.0);
	glm::ivec3 num_cells(10);
	GridCount gc(numP, min, cell_size, num_cells);

	SaveNeighborsFunctor* snfunctor = new SaveNeighborsFunctor(rad, numP, max_neighs);
	snfunctor->resetFunctor();

	glm::vec3* pos = new glm::vec3[numP];
	for (int x = 0; x < 20; x++) {
		for (int y = 0; y < 20; y++) {
			for (int z = 0; z < 20; z++) {
				int i = x + 20 * (z * 20 + y);
				if (i > numP)continue;
				pos[i] = glm::vec3(x*0.5, y*0.5, z*0.5);
			}
		}
	}

	glm::vec3* d_pos;
	hipMalloc((void**)&d_pos, numP * sizeof(glm::vec3));
	hipMemcpy(d_pos, pos, numP * sizeof(glm::vec3), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	gc.update(d_pos);
	hipDeviceSynchronize();
	gc.apply_f_frnn<SaveNeighborsFunctor>(*snfunctor, d_pos, rad);
	hipDeviceSynchronize();

	int* h_num_neighbors = new int[numP];
	int* h_neighbors = new int[numP * max_neighs];

	hipMemcpy(h_num_neighbors, snfunctor->m_num_neighbors, numP * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_neighbors, snfunctor->m_neighbors, max_neighs * numP * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(pos, d_pos, numP * sizeof(glm::vec3), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	int* res_num_neighs = new int[numP];

	for (int i = 0; i < numP; i++) {
		res_num_neighs[i] = 0;
		for (int j = 0; j < numP; j++) {
			if (i == j)continue;
			glm::vec3 dist_vec = pos[i] - pos[j];
			float dist = glm::dot(dist_vec, dist_vec);
			if (dist <= rad*rad) {
				if (i == 0)printf("i: %d j: %d pos[i]: %f %f %f pos[j]: %f %f %f dist: %f\n", i, j, pos[i].x, pos[i].y, pos[i].z, pos[j].x, pos[j].y, pos[j].z, sqrt(dist));
				res_num_neighs[i]++;
			}
		}
	}

	for (int i = 0; i < numP; i++) {

		printf("i: %d num_neighs: %d h_num_neighs: %d pos[i]: % f %f %f\n", i, res_num_neighs[i], h_num_neighbors[i], pos[i].x, pos[i].y, pos[i].z);
		for (int j = 0; j < h_num_neighbors[i]; j++) {
			double leng = glm::length(pos[h_neighbors[i * max_neighs + j]] - pos[i]);
			printf("j: %d pos: %f %f %f dist: %f\n", h_neighbors[i * max_neighs + j], pos[h_neighbors[i * max_neighs + j]].x, pos[h_neighbors[i * max_neighs + j]].y, pos[h_neighbors[i * max_neighs + j]].z, leng);
		}

		ASSERT_EQUALS(res_num_neighs[i], h_num_neighbors[i]);
	}

	delete[] res_num_neighs;
	delete[] pos;

	delete[] h_num_neighbors;
	delete[] h_neighbors;
	delete snfunctor;
	hipFree(d_pos);
}

int main() {
	init_logging();
	printf("TEST1 --------------------------------------\n\n");
	RUN(lattice_test);
	printf("\n\nTEST2 --------------------------------------\n\n");
	RUN(packed_lattice_test);
	return TEST_REPORT();
}
