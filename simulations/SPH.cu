#include "../src/rendering/Application.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <glm/glm.hpp>

#include "../src/particle_sys/particleSys.h"
#include "../src/particle_sys/sphParticleSys.h"
#include "../src/rendering/particleSysRenderer.h"

constexpr int NUM_PARTICLES = 10000;
#define TWOPI 6.2831853072

struct MyApp : public Application {
	SphParticleSys psys;
	ParticleSystemRenderer psr;


	int sim_frames_per_frame = 1;
	bool run_simulation = true;

	MyApp(int width, int height, std::string title) : Application(width, height, std::move(title)), 
		psys(NUM_PARTICLES, glm::vec3(-50.0, -50.0, -50.0), glm::vec3(50.0, 50.0, 50.0), { }),
		psr(&psys) {
		psr.radius = 21.6;
	}
	void run() {
		Timer dtTimer;
		while (!glfwWindowShouldClose(window)) {

			Timer timer;

			ImGui_ImplOpenGL3_NewFrame();
			ImGui_ImplGlfw_NewFrame();
			ImGui::NewFrame();

			ImGui::SetNextWindowSize(ImVec2(m_width/4, m_height/2)); // ensures ImGui fits the GLFW window
			ImGui::SetNextWindowPos(ImVec2(0, 0));

			ImGui::Begin("Info/Settings", NULL, ImGuiWindowFlags_NoMove | ImGuiWindowFlags_NoResize);
			ImGui::Text("Visualization");
			ImGui::Text("%.3f ms/frame (%.1f FPS)", 1000.0f / ImGui::GetIO().Framerate, ImGui::GetIO().Framerate);
			ImGui::DragInt("Simulation frames per frame", &sim_frames_per_frame, 1, 1, 20);
			ImGui::DragFloat("Zoom", &psr.zoom, 0.01, 0.5, 3.0);
			ImGui::DragFloat("Particle radius", &psr.radius, 0.1, 0.3, 50.0);
			ImGui::DragFloat("X rotation", &psr.xrot, 0.01, -TWOPI, TWOPI);
			ImGui::DragFloat("Y rotation", &psr.yrot, 0.01, -TWOPI, TWOPI);
			ImGui::DragFloat("Z rotation", &psr.zrot, 0.01, -TWOPI, TWOPI);
			ImGui::Checkbox("Show simulation box", &psr.show_cube);
			ImGui::Text("Simulation");
			ImGui::Checkbox("Running simulation", &run_simulation);
			bool sph_changed = false;
			sph_changed |= ImGui::DragFloat("Ext force strength", &psys.h_bss->ExtForce.y, 0.1, -600.0, 600.0);
			sph_changed |= ImGui::DragFloat("Radius of simulation", &psys.h_bss->KernelRadius, 0.1, 0.1, 20.0);
			sph_changed |= ImGui::DragFloat("Viscosity", &psys.h_bss->Viscosity, 0.1, 0.1, 100.0);
			sph_changed |= ImGui::DragFloat("Rho 0",   &psys.h_bss->RestDensity, 0.1, 0.1, 100.0);
			sph_changed |= ImGui::DragFloat("Gas constant (k)", &psys.h_bss->GasConst, 0.1, 0.1, 100.0);
			sph_changed |= ImGui::DragFloat("Particle mass", &psys.h_bss->PartMass, 0.1, 0.1, 100.0);
			sph_changed |= ImGui::DragFloat("Col restitution", &psys.h_bss->ColRestitution, 0.1, 0.1, 100.0);

			ImGui::End();

			ImGui::Render();

			if (sph_changed) {
				hipMemcpy(psys.d_bss, psys.h_bss, sizeof(sph_sim_settings), hipMemcpyHostToDevice);
			}

			LOG_TIMING("ImGui tab setting: {} ms", timer.swap_time());

			float dt = dtTimer.swap_time(); // dt is in ms
			for (int c = 0; c < sim_frames_per_frame && run_simulation; c++) {
				psys.update(dt / 10000.0);
				//printf("timestep\n");
			}

			LOG_TIMING("Particle update time: {} ms", timer.swap_time());

			// clear
			glClear(GL_COLOR_BUFFER_BIT);
			glClearColor(0.7, 0.7, 0.7, 0.0);
			glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
			GLCHECKERR();

			psr.renderps(m_width,m_height);
			GLCHECKERR();

			ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

			LOG_TIMING("Render time: {} ms", timer.swap_time());

			glfwSwapBuffers(window);
			glfwPollEvents();

			dispatch_events();

			LOG_TIMING("Event handling + swaping buffers time: {} ms", timer.swap_time());

		}
	}
};


int main(){
	MyApp app(1200,800,"Main Example");
	app.run();
}